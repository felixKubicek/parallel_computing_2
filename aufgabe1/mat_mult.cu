#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h> 
#include <stdbool.h>
#include <assert.h>
#include "../aufgabe2/ca_common.h"

#define BLOCK_SIZE 32
#define TILE_WIDTH BLOCK_SIZE

#define M(m, row, col) m.elements[(row) * m.cols + (col)]
#define SIZE(m) m.cols * m.rows * sizeof(m_cell)

#ifdef TILING
#define MULT_KERNEL mat_mult_tiling_kernel
#else
#define MULT_KERNEL mat_mult_kernel
#endif

#define m_cell_fms "%d"
typedef int m_cell;

typedef struct { 
    int rows;
    int cols; 
    m_cell* elements;
} Matrix;

bool matrix_equal(const Matrix a, const Matrix b);
void mat_mult(const Matrix, const Matrix, const Matrix);
void print_matrix(const Matrix);
__global__ void mat_mult_kernel(const Matrix a, const Matrix b, const Matrix c);
__global__ void mat_mult_tiling_kernel(const Matrix a, const Matrix b, const Matrix c);
bool set_cache_config(int cache_config, const char ** cache_config_str);
void mat_mult_init(int argc, char** argv, int *n, int *cache_config, const char ** cache_config_str);

int main(int argc, char** argv)
{
    int n, cache_config;
    const char *cache_config_str;
    
    mat_mult_init(argc, argv, &n, &cache_config, &cache_config_str);

    Matrix a, b, c;
    Matrix d_a, d_b, d_c;
 
    a.cols = a.rows = n;
    b.cols = b.rows = n;

    c.rows = a.rows;
    c.cols = b.cols;

    MALLOC_ERROR_CHECK(a.elements = (m_cell*) malloc(SIZE(a)));
    MALLOC_ERROR_CHECK(b.elements = (m_cell*) malloc(SIZE(b)));
    MALLOC_ERROR_CHECK(c.elements = (m_cell*) malloc(SIZE(c)));

    // c_host matrix calculated locally (compared with c matrix for validation) 
    Matrix c_host;
    c_host.rows = c.rows;
    c_host.cols = c.cols;
    MALLOC_ERROR_CHECK(c_host.elements = (m_cell*) malloc(SIZE(c_host)));

    int row;
    int col;
    for (row = 0; row < n; row++)
    {
        for(col = 0; col < n; col++)
        {
            M(a, row, col) = row * n + col;
            M(b, col, row) = row * n + col;
        }
    }

    //print_matrix(a);
    //print_matrix(b);

    d_a.cols = a.cols;
    d_a.rows = a.rows;
    hipMalloc(&d_a.elements,SIZE(d_a));
    hipMemcpy(d_a.elements, a.elements, SIZE(d_a), hipMemcpyHostToDevice);

    d_b.cols = b.cols;
    d_b.rows = b.rows;
    hipMalloc(&d_b.elements, SIZE(d_b));
    hipMemcpy(d_b.elements, b.elements, SIZE(d_b), hipMemcpyHostToDevice);

    d_c.cols = c.cols;
    d_c.rows = c.rows;
    hipMalloc(&d_c.elements, SIZE(d_c));

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    dim3 dimGrid(c.cols / dimBlock.x, c.rows / dimBlock.y);

        
    TIME_GET(start);
    MULT_KERNEL<<<dimGrid, dimBlock>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();
    TIME_GET(stop);
    
    double kernel_time = TIME_DIFF(start, stop);
   
    CUDA_ERROR_CHECK(hipPeekAtLastError());

    hipMemcpy(c.elements, d_c.elements, SIZE(d_c), hipMemcpyDeviceToHost);

    //print_matrix(c);

    mat_mult(a ,b ,c_host);
    // validate c (kernel result matrix) with c_host matrix
    bool valid_result = matrix_equal(c, c_host);
  
    // free memory
    free(a.elements);
    free(b.elements);
    free(c.elements);
    free(c_host.elements);

    hipFree(d_a.elements);
    hipFree(d_b.elements);
    hipFree(d_c.elements);

    if (valid_result)
    {
        printf("{ \"valid\": true, \"n\": %d, \"kernel_time\": %.9f, \"cache_config\": \"%s\", \"kernel\": \"%s\"}\n", n, kernel_time, cache_config_str, STR(MULT_KERNEL));
        return EXIT_SUCCESS;
    }
    else
    { 
        printf("{\"valid\": false}\n");
        return EXIT_FAILURE;
    }
}

void mat_mult_init(int argc, char** argv, int *n, int *cache_config, const char ** cache_config_str)
{
	assert(argc == 3);

	*n = atoi(argv[1]);
	*cache_config = atoi(argv[2]);

	assert((*n > 0) && (*n % BLOCK_SIZE == 0));
        assert(set_cache_config(*cache_config, cache_config_str));
}

bool set_cache_config(int cache_config, const char ** cache_config_str)
{
    bool success = true;
    
    switch(cache_config)
    {
        case hipFuncCachePreferNone:
            CUDA_ERROR_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(MULT_KERNEL), hipFuncCachePreferNone));
            *cache_config_str = "prefer_none";
            break;
        case hipFuncCachePreferShared:
            CUDA_ERROR_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(MULT_KERNEL), hipFuncCachePreferShared));
            *cache_config_str = "prefer_shared";
            break; 
        case hipFuncCachePreferL1:
            CUDA_ERROR_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(MULT_KERNEL), hipFuncCachePreferL1));
            *cache_config_str = "prefer_L1";
            break; 
        case hipFuncCachePreferEqual:
            CUDA_ERROR_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(MULT_KERNEL), hipFuncCachePreferEqual));
            *cache_config_str = "prefer_equal";
            break;
        default :
            *cache_config_str = "undefined";
            success = false;
    }

    return success;
}

bool matrix_equal(const Matrix a, const Matrix b)
{
    if (a.cols != b.cols || a.rows != b.rows)
    {
        return false;
    } 
    
    int row;
    int col;
    for (row = 0; row < a.rows; row++)
    {
        for(col = 0; col < a.cols; col++)
        {
            if (M(a, row, col) != M(b, row, col))
            {
                return false;
            } 
        }

    }
    
    return true;
}

void mat_mult(const Matrix a, const Matrix b, const Matrix c)
{
    int row;
    int col;
    for (row = 0; row < c.rows; row++)
    {
        for(col = 0; col < c.cols; col++)
        {
            int inner;
            m_cell sum = 0;
            for (inner = 0; inner < a.cols; inner++)
            {
                sum += M(a, row, inner) * M(b, inner, col);
            }
            M(c, row, col) = sum; 
        }
    }
}

__global__ void mat_mult_kernel(const Matrix a, const Matrix b, const Matrix c) 
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int inner;
    m_cell sum = 0;
    for (inner = 0; inner < a.cols; inner++)
    {
        sum += M(a, row, inner) * M(b, inner, col);
    }
    M(c, row, col) = sum; 
}

__global__ void mat_mult_tiling_kernel(const Matrix a, const Matrix b, const Matrix c)
{
    __shared__ m_cell a_ds [TILE_WIDTH][TILE_WIDTH];
    __shared__ m_cell b_ds [TILE_WIDTH][TILE_WIDTH];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    m_cell cval = 0;

    int t;
    for (t = 0; t < a.cols/TILE_WIDTH; t++)
    {
        a_ds[ty][tx] = M(a, row, t*TILE_WIDTH+tx);
        b_ds[ty][tx] = M(b, t*TILE_WIDTH + ty, col);
        
        __syncthreads();
        
        int i; 
        for(i=0; i < TILE_WIDTH; i++)
        {
            cval += a_ds[ty][i] * b_ds[i][tx];
        }
        
        __syncthreads();
    }

    M(c, row, col) = cval;
}

void print_matrix(const Matrix m)
{
    int row;
    int col;
    for (row = 0; row < m.rows; row++)
    {
        for(col = 0; col < m.cols; col++)
        {
            printf("\t"m_cell_fms, M(m, row, col));
        }
        printf("\n");
    }
    printf("\n\n");
}

