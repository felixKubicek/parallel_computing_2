#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h> 
#include "ca_common.h"

#define BLOCK_SIZE 32
#define N BLOCK_SIZE

#define TILE_WIDTH BLOCK_SIZE

#define m_cell_fms "%d"

#define M(m, row, col) m.elements[row * m.rows + col]
#define SIZE(m) m.cols * m.rows * sizeof(m_cell)

#define CUDA_ERROR_CHECK(x)\
    do {hipError_t last_err = (x);\
        if (last_err != hipSuccess)\
                {fprintf(stderr ,"%s:%u: CUDA error: %s\n", __FILE__, __LINE__, hipGetErrorString ( last_err )); exit(EXIT_FAILURE); }\
    } while (false)

#define MALLOC_ERROR_CHECK(x)\
        do {\
            if ( (x) == NULL)\
                {fprintf(stderr ,"%s:%u: malloc error!\n", __FILE__, __LINE__); exit (EXIT_FAILURE); }\
        } while (false)

typedef int m_cell;

typedef struct { 
    int rows;
    int cols; 
    m_cell* elements;
} Matrix;


void mat_mult(const Matrix, const Matrix, const Matrix);
void print_matrix(const Matrix);
__global__ void mat_mult_kernel(const Matrix, const Matrix, const Matrix);
__global__ void mat_mult_tiling_kernel(const Matrix, const Matrix, const Matrix);


int main(void)
{
    Matrix a, b, c;
    Matrix d_a, d_b, d_c;

    a.cols = a.rows = N;
    b.cols = b.rows = N;

    c.rows = a.rows;
    c.cols = b.cols;

    MALLOC_ERROR_CHECK(a.elements = (m_cell*) malloc(SIZE(a)));
    MALLOC_ERROR_CHECK(b.elements = (m_cell*) malloc(SIZE(b)));
    MALLOC_ERROR_CHECK(c.elements = (m_cell*) malloc(SIZE(c)));

    int row;
    int col;
    for (row = 0; row < N; row++)
    {
        for(col = 0; col < N; col++)
        {
            M(a, row, col) = row * N + col;
            M(b, col, row) = row * N + col;
        }
    }

    d_a.cols = a.cols;
    d_a.rows = a.rows;
    hipMalloc(&d_a.elements,SIZE(d_a));
    hipMemcpy(d_a.elements, a.elements, SIZE(d_a), hipMemcpyHostToDevice);

    d_b.cols = b.cols;
    d_b.rows = b.rows;
    hipMalloc(&d_b.elements, SIZE(d_b));
    hipMemcpy(d_b.elements, b.elements, SIZE(d_b), hipMemcpyHostToDevice);

    d_c.cols = c.cols;
    d_c.rows = c.rows;
    hipMalloc(&d_c.elements, SIZE(d_c));

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    dim3 dimGrid(c.cols / dimBlock.x, c.rows / dimBlock.y);
    
    TIME_GET(start);
    mat_mult_kernel<<<dimGrid, dimBlock>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();
    TIME_GET(stop);
    
    CUDA_ERROR_CHECK(hipPeekAtLastError());

    hipMemcpy(c.elements, d_c.elements, SIZE(d_c), hipMemcpyDeviceToHost);

    print_matrix(a);
    print_matrix(b);
    print_matrix(c);
    
    // free memory
    free(a.elements);
    free(b.elements);
    free(c.elements);

    hipFree(d_a.elements);
    hipFree(d_b.elements);
    hipFree(d_c.elements);

    return EXIT_SUCCESS;
}


__global__ void mat_mult_kernel(const Matrix a, const Matrix b, const Matrix c) 
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int inner;
    m_cell sum = 0;
    for (inner = 0; inner < a.cols; inner++)
    {
        sum += M(a, row, inner) * M(b, inner, col);
    }
    M(c, row, col) = sum; 
}


__global__ void mat_mult_tiling_kernel(const Matrix a, const Matrix b, const Matrix c)
{
    __shared__ m_cell a_ds [TILE_WIDTH][TILE_WIDTH];
    __shared__ m_cell b_ds [TILE_WIDTH][TILE_WIDTH];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    m_cell cval = 0;

    int t;
    for (t = 0; t < a.cols/TILE_WIDTH; t++)
    {
        a_ds[ty][tx] = M(a, row, t*TILE_WIDTH+tx);
        b_ds[ty][tx] = M(b, t*TILE_WIDTH + ty, col);
        
        __syncthreads();
        
        int i; 
        for(i=0; i < TILE_WIDTH; i++)
        {
            cval += a_ds[ty][i] * b_ds[i][tx];
        }
        
        // synchronization here required?
        __syncthreads();
    }

    M(c, row, col) = cval;
}


void mat_mult(const Matrix a, const Matrix b, const Matrix c)
{
    int row;
    int col;
    for (row = 0; row < c.rows; row++)
    {
        for(col = 0; col < c.cols; col++)
        {
            int inner;
            m_cell sum = 0;
            for (inner = 0; inner < a.cols; inner++)
            {
                sum += M(a, row, inner) * M(b, inner, col);
            }
            M(c, row, col) = sum; 
        }
    }
}


void print_matrix(const Matrix m)
{
    int row;
    int col;
    for (row = 0; row < m.rows; row++)
    {
        for(col = 0; col < m.cols; col++)
        {
            printf("\t"m_cell_fms, M(m, row, col));
        }
        printf("\n");
    }
    printf("\n\n");
}
